#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void matwrite(int N, float *A){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  
  if (i < N && j < N){
    A[i + j * N] = 1.f; // column major 
    A[j + i * N] = 1.f; // row major
  }
}

int main(int argc, char * argv[]){

  int N = 4096;
  int blockSize = 32;
  if (argc > 1){
    N = atoi(argv[1]);
    blockSize = atoi(argv[2]);
  }
  printf("N = %d, blockSize = %d\n", N, blockSize);

  float * A = new float[N * N];
  for (int i = 0; i < N * N; ++i){
    A[i] = 0.f; 
  }

  // allocate memory and copy to the GPU
  float * d_A;
  int size_A = N * N * sizeof(float);
  hipMalloc((void **) &d_A, size_A);
  
  // copy memory over to the GPU
  hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);

  // Next largest multiple of blockSize
  int numBlocks = (N + blockSize - 1) / blockSize; 
  printf("N = %d, numBlocks * blockSize = %d\n", N, numBlocks * blockSize);

  dim3 blockDims(blockSize, blockSize);
  dim3 gridDims(numBlocks, numBlocks);

#if 1
  float time;
  float min_time = 1e6;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  for (int i = 0; i < 10; ++i){
    hipEventRecord(start, 0);
    matwrite <<< gridDims, blockDims >>> (N, d_A);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    if (time < min_time)
        min_time = time;
  }
  
  printf("Time to run kernel: %6.2f ms.\n", min_time);
  
#endif

  return 0;
}
